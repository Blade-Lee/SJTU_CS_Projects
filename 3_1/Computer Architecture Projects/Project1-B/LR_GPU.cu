#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

#include <thrust\transform.h>
#include <thrust\transform_reduce.h>
#include <thrust\device_ptr.h>
#include <thrust\device_vector.h>
#include <thrust\host_vector.h>
#include <thrust\functional.h>
#include <thrust\iterator\counting_iterator.h>
#include <thrust\sequence.h>

#include "LR_GPU_Functors.cu"

//DLL exports
extern "C" __declspec(dllexport) int __cdecl Learn(float*, float*, unsigned int, unsigned int, unsigned int, float, float, float*, float*, float*);
extern "C" __declspec(dllexport) int __cdecl Predict(float*, unsigned int, unsigned int, float*, float *, float *, float *);


//
//This method does mean normalization
//
void NormalizeFeaturesByMeanAndStd(unsigned int trainingDataCount, float * d_trainingData, thrust::device_vector<float> dv_mean, thrust::device_vector<float> dv_std)
{
	//Calculate mean norm: (x - mean) / std
	//featureCount == 8
	unsigned int featureCount = dv_mean.size();
	float * dvp_Mean = thrust::raw_pointer_cast( &dv_mean[0] );
	float * dvp_Std = thrust::raw_pointer_cast( &dv_std[0] );
	FeatureNormalizationgFunctor featureNormalizationgFunctor(dvp_Mean, dvp_Std, featureCount); 
	thrust::device_ptr<float> dvp_trainingData(d_trainingData); 
	thrust::transform(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(trainingDataCount * featureCount), dvp_trainingData, dvp_trainingData, featureNormalizationgFunctor);
}

//
//This method calculates mean, standard deviation and does mean normalization
//
void NormalizeFeatures(unsigned int featureCount, unsigned int trainingDataCount, float * d_trainingData, float * meanResult, float * stdResult)
{
	//featureCount == 8

	//Calculate the mean. One thread per feature.
	thrust::device_vector<float> dv_mean(featureCount,0);
	MeanFunctor meanFunctor(d_trainingData, trainingDataCount, featureCount); 
	thrust::transform(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(featureCount), dv_mean.begin(), meanFunctor);

	//Calculate the standard deviation. One thread per feature.
	thrust::device_vector<float> dv_std(featureCount,0);
	STDFunctor stdFunctor(d_trainingData, trainingDataCount, featureCount); 
	thrust::transform(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(featureCount), dv_mean.begin(), dv_std.begin(), stdFunctor);

	//Calculate mean norm: (x - mean) / std
	NormalizeFeaturesByMeanAndStd(trainingDataCount, d_trainingData, dv_mean, dv_std);

	thrust::copy(dv_mean.begin(), dv_mean.end(), meanResult);
	thrust::copy(dv_std.begin(), dv_std.end(), stdResult);
}

void AddBiasTerm(float * inputData, float * outputData, int dataCount, int featureCount)
{
	//transfer the trainindata by adding also the bias term
	//featureCount == 8
	for(int i = 0; i < dataCount; i++)
	{
		//all the first feature is 1
		outputData[i * featureCount] = 1;
		for(int f = 1; f < featureCount; f++)
			outputData[i * featureCount + f] = inputData[(i * (featureCount - 1)) + (f-1)];
	}

}

#define IsValidNumber(x)  (x == x && x <= DBL_MAX && x >= -DBL_MAX)

//
//Learn the hypothesis for the given data
//
extern int Learn(float* trainingData, float * labelData, unsigned int featureCount, unsigned int trainingDataCount, unsigned int gdIterationCount, float learningRate, float regularizationParam, float * result, float * meanResult, float * stdResult)
{
	//featureCount == 7 -> == 8
	featureCount++;
	//allcate host memory
	thrust::host_vector<float> hv_hypothesis(featureCount, 0);
	thrust::host_vector<float> hv_trainingData(trainingDataCount * featureCount);
	thrust::host_vector<float> hv_labelData(labelData, labelData + trainingDataCount);
	//transfer the trainindata by adding also the bias term
	AddBiasTerm(trainingData, &hv_trainingData[0], trainingDataCount, featureCount);
	
	//allocate device vector
	thrust::device_vector<float> dv_hypothesis = hv_hypothesis;
	thrust::device_vector<float> dv_trainingData = hv_trainingData;
	thrust::device_vector<float> dv_labelData = hv_labelData;
	thrust::device_vector<float> dv_costData(trainingDataCount, 0);
	//Get device vector pointers
	float * pdv_hypothesis = thrust::raw_pointer_cast( &dv_hypothesis[0] );
	float * pdv_trainingData = thrust::raw_pointer_cast( &dv_trainingData[0] );
	float * pdv_costData = thrust::raw_pointer_cast( &dv_costData[0] );
	
	//Normalize the features
	NormalizeFeatures(featureCount, trainingDataCount, pdv_trainingData, meanResult, stdResult);

	TrainFunctor tf(pdv_trainingData, pdv_hypothesis, featureCount);
	TrainFunctor2 tf2(pdv_costData, pdv_trainingData, featureCount);
	//run gdIterationCount of gradient descent iterations
	for(int i = 0; i < gdIterationCount; i++)
	{
		thrust::transform(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(trainingDataCount),  dv_labelData.begin(), dv_costData.begin(), tf);

		//calculate gradient descent iterations
		for(int featureNumber = 0; featureNumber < featureCount; featureNumber++) 
		{
			tf2.SetFeatureNumber(featureNumber);
			float totalCost = thrust::transform_reduce(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(trainingDataCount),  tf2, 0.0f, thrust::plus<float>());
			if (!IsValidNumber(totalCost))
			{
				i = gdIterationCount;
				break;
			}
			float regularizationTerm = 1 - (learningRate * (regularizationParam / trainingDataCount));
			hv_hypothesis[featureNumber] = (hv_hypothesis[featureNumber] * regularizationTerm) -  learningRate * (totalCost / trainingDataCount);
		}
		
		//Copy the theta back to the device vector
		dv_hypothesis = hv_hypothesis;
	}

	//copy the hypothesis into the result buffer
	thrust::copy(hv_hypothesis.begin(), hv_hypothesis.end(), result);

	return 0;
}

//
//makes prediction for the given test data based on the hypothesis. Also applies feature normalization.
//
extern int Predict(float* testData, unsigned int featureCount, unsigned int testDataCount, float* hypothesis, float * mean, float * std, float * result)
{
	//featureCount == 7 -> == 8
	featureCount++;
	thrust::host_vector<float> hv_testData(testDataCount * featureCount);
	AddBiasTerm(testData, &hv_testData[0], testDataCount, featureCount);
	
	//Allocate device memory
	thrust::device_vector<float> dv_hypothesis(hypothesis, hypothesis + featureCount);
	thrust::device_vector<float> dv_testData = hv_testData;
	thrust::device_vector<float> dv_result(testDataCount);
	thrust::device_vector<float> dv_mean(mean, mean + featureCount);
	thrust::device_vector<float> dv_std(std, std + featureCount);

	//Normalize features
	float * pdv_hypothesis = thrust::raw_pointer_cast( &dv_hypothesis[0] );
	float * pdv_testData = thrust::raw_pointer_cast( &dv_testData[0] );
	NormalizeFeaturesByMeanAndStd(testDataCount, pdv_testData, dv_mean, dv_std);

	//Predict
	PredictFunctor predictFunctor(pdv_testData, pdv_hypothesis, featureCount);
	thrust::transform(thrust::counting_iterator<int>(0), thrust::counting_iterator<int>(testDataCount), dv_result.begin(), predictFunctor);

	//copy the result from device memory into the result buffer
	thrust::copy(dv_result.begin(), dv_result.end(), result);

	return 0;
}



