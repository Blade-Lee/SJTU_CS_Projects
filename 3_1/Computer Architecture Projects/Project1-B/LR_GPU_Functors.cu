#include "hip/hip_runtime.h"
#include <thrust\functional.h>

//
//Calculates the mean for a column major ordered matrix
//
struct MeanFunctor : public thrust::unary_function<int, float>
{
	float * trainingData;
	unsigned int trainingDataCount;
	unsigned int featureCount;

	MeanFunctor(float * _trainingData, unsigned int _trainingDataCount, unsigned int _featureCount) : trainingData(_trainingData), trainingDataCount(_trainingDataCount), featureCount(_featureCount)
	{}

	//tid:[0 , featureCount)
  __host__ __device__
  float operator()(int tid)
  {
	  float sum = 0;
	  for(int i = 0; i < trainingDataCount; i++)
		  sum += trainingData[featureCount * i + tid];
	  
	  return sum / trainingDataCount;
  }
};


//
//Calculates the standard deviation for a column major ordered matrix
//
struct STDFunctor : public thrust::binary_function<int, float, float>
{
	float * trainingData;
	unsigned int trainingDataCount;
	unsigned int featureCount;

	STDFunctor(float * _trainingData, unsigned int _trainingDataCount, unsigned int _featureCount ) : trainingData(_trainingData), trainingDataCount(_trainingDataCount), featureCount(_featureCount)
	{}

  __host__ __device__
  float operator()(int tid, float meanValue)
  {
	  float sum = 0;
	  for(int i = 0; i < trainingDataCount; i++)
		  sum += powf(trainingData[featureCount * i + tid] - meanValue, 2.0);
	  
	  return sqrtf(sum / (float)trainingDataCount);
  }
};

//
//Runs the first part of the training
//
struct TrainFunctor : public thrust::binary_function<int, float, float>
{
	float * trainingData;
	float * hypothesis;
	unsigned int featureCount;

	TrainFunctor(float * _trainingData, float * _hypothesis, int _featureCount) : trainingData(_trainingData), hypothesis(_hypothesis), featureCount(_featureCount)
	{}

  __host__ __device__
  float operator()(int tid, float labelData)
  {
	//Please complete the code!
	  //tid: [0 , trainingDataCount)
	  float sum = 0;
	  for (int i = 0; i < featureCount; ++i){
			sum += hypothesis[i] * trainingData[featureCount * tid + i];
	  }
	  return sum - labelData;
  }
};

//
//Runs the second part of the training
//
struct TrainFunctor2 : public thrust::unary_function<int, float>
{
	float * trainingData;
	float * costData;
	unsigned int featureNumber;
	unsigned int featureCount;

	TrainFunctor2(float * _costData, float * _trainingData, unsigned int _featureCount) : costData(_costData), trainingData(_trainingData), featureCount(_featureCount), featureNumber(0)
	{}

	void SetFeatureNumber(unsigned int value)
	{
		featureNumber = value;
	}

  __host__ __device__
  float operator()(int tid)
  {
	  return costData[tid] * trainingData[tid * featureCount + featureNumber];
  }
};
//
//Applies feature normalization algorithm to the data. (data - mean) / standard deviation
//
struct FeatureNormalizationgFunctor : public thrust::binary_function<int, float, float>
{
	float * meanValue;
	float * stdValue;
	unsigned int featureCount;

	FeatureNormalizationgFunctor(float * _meanValue, float * _stdValue, unsigned int _featureCount) : meanValue(_meanValue), stdValue(_stdValue), featureCount(_featureCount)
	{}

  __host__ __device__
  float operator()(int tid, float trainingData)
  {
	//please complete the code!
	  //tid: [0 , trainingDataCount * featureCount)
	  int index = tid % featureCount;
	  if (index != 0)
		return (trainingData - meanValue[index]) / stdValue[index];
	  else
		return trainingData;
  }
};

//
//Applies the hypothesis to the test data
//
struct PredictFunctor : public thrust::unary_function<int, float>
{
	float * testData;
	float * hypothesis;
	unsigned int featureCount;

	PredictFunctor(float * _testData, float * _hypothesis, unsigned int _featureCount) : testData(_testData), hypothesis(_hypothesis), featureCount(_featureCount)
	{}

	//tid: [0 , testDataCount)
  __host__ __device__
  float operator()(int tid)
  {
	  //Please complete the code!
	  int sum = 0;
	  sum += hypothesis[0];
	  for (int i = 1; i < featureCount; ++i){
		  sum += hypothesis[i] * testData[tid * featureCount + i];
	  }
	  return sum;
  }
};